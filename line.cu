#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <vector>
#include <cstring>

/**
 * Kernel fusionnant le passage en niveaux de gris et la détection de contours.
 */
__global__ void grayscale_line_shared( unsigned char * rgb, unsigned char * s, std::size_t cols, std::size_t rows ) {
  auto i = blockIdx.x * (blockDim.x-2) + threadIdx.x;
  auto j = blockIdx.y * (blockDim.y-2) + threadIdx.y;

  auto li = threadIdx.x;
  auto lj = threadIdx.y;

  auto w = blockDim.x;
  auto h = blockDim.y;

  extern __shared__ unsigned char sh[];

  if( i < cols && j < rows ) {
    sh[ lj * w + li ] = (
			 307 * rgb[ 3 * ( j * cols + i ) ]
			 + 604 * rgb[ 3 * ( j * cols + i ) + 1 ]
			 + 113 * rgb[  3 * ( j * cols + i ) + 2 ]
			 ) >> 10;
  }

  /**
   * Il faut synchroniser tous les warps (threads) du bloc pour être certain que le niveau de gris est calculé
   * par tous les threads du bloc avant de pouvoir accéder aux données des pixels voisins.
   */
  __syncthreads();
 
  if( i < cols -1 && j < rows-1 && li > 0 && li < (w-1) && lj > 0 && lj < (h-1) )
  {
      
    // Quarante cinq degrees
    auto qc = 
        - sh[((lj - 1) * w + li - 1)] - sh[((lj -1) * w + li)] + 2 * sh[((lj -1) * w + li + 1)]
        - sh[(lj * w + li - 1)] + 2 * sh[(lj * w + li)] - sh[(lj * w + li + 1)]
        + 2 * sh[((lj + 1) * w + li - 1)] - sh[((lj + 1) * w + li)] - sh[((lj + 1) * w + li + 1)];
                
   // Cent trente cinq degrees
   /* auto ctc = 
        2 * sh[((lj - 1) * w + li - 1)] - sh[((lj -1) * w + li)] - sh[((lj -1) * w + li + 1)]
        - sh[(lj * w + li - 1)] + 2 * sh[(lj * w + li)] - sh[(lj * w + li + 1)]
        - sh[((lj + 1) * w + li - 1)] - sh[((lj + 1) * w + li)] + 2 * sh[((lj + 1) * w + li + 1)];
	*/
        
    qc = qc > 255 ? 255 : qc;
    qc = qc < 0 ? 0 : qc;
    
    s[ j * cols + i ] = qc;
  }
}


int main()
{
  cv::Mat m_in = cv::imread("in2.jpg", cv::IMREAD_UNCHANGED );

  //auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;

  //std::vector< unsigned char > g( rows * cols );
  // Allocation de l'image de sortie en RAM côté CPU.
  unsigned char * g = nullptr;
  hipHostMalloc( &g, rows * cols );
  cv::Mat m_out( rows, cols, CV_8UC1, g );

  // Copie de l'image en entrée dans une mémoire dite "pinned" de manière à accélérer les transferts.
  // OpenCV alloue la mémoire en interne lors de la décompression de l'image donc soit sans doute avec
  // un malloc standard.
  unsigned char * rgb = nullptr;
  hipHostMalloc( &rgb, 3 * rows * cols );
  
  std::memcpy( rgb, m_in.data, 3 * rows * cols );

  unsigned char * rgb_d;
  unsigned char * g_d;
  unsigned char * s_d;

  hipMalloc( &rgb_d, 3 * rows * cols );
  hipMalloc( &g_d, rows * cols );
  hipMalloc( &s_d, rows * cols );

  hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );

  dim3 block( 64, 8 );
  dim3 grid0( ( cols - 1) / block.x + 1 , ( rows - 1 ) / block.y + 1 );
  /**
   * Pour la version shared il faut faire superposer les blocs de 2 pixels
   * pour ne pas avoir de bandes non calculées autour des blocs
   * on crée donc plus de blocs.
   */
  dim3 grid1( ( cols - 1) / (block.x-2) + 1 , ( rows - 1 ) / (block.y-2) + 1 );
    
  hipEvent_t start, stop;

  hipEventCreate( &start );
  hipEventCreate( &stop );

  // Mesure du temps de calcul du kernel uniquement.
  hipEventRecord( start );

  // Version fusionnée.
  grayscale_line_shared<<< grid1, block, block.x * block.y >>>( rgb_d, s_d, cols, rows );
  
  // Vérification des erreurs
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
      printf("Erreur CUDA: %s\n", hipGetErrorString(error));
  }

  hipEventRecord( stop );
  
  hipMemcpy( g, s_d, rows * cols, hipMemcpyDeviceToHost );

  hipEventSynchronize( stop );
  float duration;
  hipEventElapsedTime( &duration, start, stop );
  std::cout << "time=" << duration << "ms" << std::endl;

  hipEventDestroy(start);
  hipEventDestroy(stop);

  cv::imwrite( "out_line_cu.jpg", m_out );

  hipFree( rgb_d);
  hipFree( g_d);
  hipFree( s_d);

  hipHostFree( g );
  hipHostFree( rgb );
  
  return 0;
}
